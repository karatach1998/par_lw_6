#include "hip/hip_runtime.h"
#include <stdio.h>
#include <argp.h>


#define N 8


enum { H, A, B };


__global__
void kernel(float* data)
{
    __constant__ float cs; // constants
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float x = cs[H] * idx;

    float arg = absf((cs[A] * x + cs[B]) * x * x - cs[A] * cs[B]);
    data[idx] = powf(sinf(arg), 2) / sqrtf(arg);
}


struct config
{
    enum execution_side { DEVICE, HOST } execution_side;
    unsigned k;
};


static int parse_arg(int key, const char* arg, struct argp_state* state)
{
    struct config* config = state->input;
    switch (key)
    {
        case 'c': config->execution_side = HOST; break;
        case 'g': config->execution_side = DEVICE; break;
        case 'k': config->k = atoi(arg); break;
        default: argp_failure(state, 0, EINVAL, "Invalid argument."); return EINVAL;
    }
    return 0;
}


struct argp_option options[] = {
    {"cpu", 'c', 0, 0, "Execute on CPU."},
    {"gpu", 'g', 0, 0, "Execute on GPU."},
    {0, 'k', "N", 0, "K-value."},
    { 0 }
};

struct argp argp = {options, parse_arg};


int main(int argc, char* argv[])
{
    struct config config = {};

    argp_parse(&argp, argc, argv, 0, 0, &config);

    float a[config.k];
    float local_constants[3];

    local_constants[H] = N / config.k;
    local_constants[A] = N;
    local_constants[B] = N * 2;
    hipMemcpyToSymbol(HIP_SYMBOL(cs), local_constants, sizeof local_constants);

    float* dev_a = NULL;
    hipEvent_t start, stop;
    float elapsed_time;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipMalloc((void**) &dev_a, sizeof a);

    hipEventRecord(&start);
    kernel<<dim3(config.k / 512), dim3(512)>>>(dev_a);
    hipEventRecord(&stop);
    cudaEvnetElapsedTime(&elapsed_time, start, stop);

    hipMemcpy(a, dev_a, sizeof a, hipMemcpyDeviceToHost);
    hipFree(dev_a);

    printf("Elapsed time: %f\n", elapsed_time);
    if (config->flags & PRINT_FLAG) {
        printf("*** Computation results ***\n");
        for (unsigned i = 0; i < config.k; ++i) {
            printf("f(%f) = %f\n", local_constants[H] * i, a[i]);
        }
    }

    return 0;
}
